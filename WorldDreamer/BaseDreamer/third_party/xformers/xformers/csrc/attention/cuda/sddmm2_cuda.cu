#include "hip/hip_runtime.h"
// taken from
// https://github.com/hgyhungry/ge-spmm/blob/master/pytorch-custom/sddmm.cu with
// slight modifications to add batch support
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/types.h>
#include "../computeUtil.h"

namespace ge_spmm {

template <typename T>
__global__ void sddmmCOO4Scale(
    int S_mrows,
    int D_kcols,
    int S_nrows,
    const unsigned long Size,
    int* S_cooRowInd,
    int* S_cooColInd,
    T* D1_dnVal_,
    T* D2_dnVal_,
    T* O_cooVal_) {
  int eid = (blockIdx.x << 4) + (threadIdx.y << 2);
  int cid = (threadIdx.x << 2);

  int batch_id = blockIdx.y;
  int batch_offset_1 = batch_id * S_mrows * D_kcols;
  int batch_offset_2 = batch_id * S_nrows * D_kcols;
  int batch_offset_out = batch_id * Size;

  T* D1_dnVal = D1_dnVal_ + batch_offset_1;
  T* D2_dnVal = D2_dnVal_ + batch_offset_2;
  T* O_cooVal = O_cooVal_ + batch_offset_out;

  if (blockIdx.x < Size / 16) {
    T multi[4] = {0, 0, 0, 0};
    int offset1[4], offset2[4];
    float4 D1tmp[4], D2tmp[4];
    Load<int4, int>(offset1, S_cooRowInd, eid);
    Load<int4, int>(offset2, S_cooColInd, eid);
    selfMulConst4<int>(offset1, D_kcols);
    selfMulConst4<int>(offset2, D_kcols);

    for (int i = 0; i < (D_kcols >> 5); i++) {
      Load4<float4, float>(D1tmp, D1_dnVal, offset1, cid);
      Load4<float4, float>(D2tmp, D2_dnVal, offset2, cid);
      vec4Dot4<float4, float>(multi, D1tmp, D2tmp);
      cid += 32;
    }
    int res = D_kcols & 31;
    if (res) {
      int cid2 = threadIdx.x + D_kcols - res;
      float D1[4] = {0, 0, 0, 0}, D2[4] = {0, 0, 0, 0};
      for (int i = 0; i < res / 8 + 1; i++) {
        if (i * 8 + threadIdx.x < res) {
          Load4<float, float>(D1, D1_dnVal, offset1, cid2);
          Load4<float, float>(D2, D2_dnVal, offset2, cid2);
          Dot4<float>(multi, D1, D2);
          cid2 += 8;
        }
      }
    }
    AllReduce4<float>(multi, 4, 32);
    if (threadIdx.x == 0) {
      Store<float4, float>(O_cooVal, multi, eid);
    }
  } else // Dynamic parrallel?
  {
    eid = Size - (Size & 15) + (blockIdx.x - (Size / 16));
    int offset1 = S_cooRowInd[eid] * D_kcols;
    int offset2 = S_cooColInd[eid] * D_kcols;
    T multi = 0;
    int off1 = cid = threadIdx.x + (threadIdx.y << 3);
    float D1tmp0, D2tmp0;
    for (int cc = 0; cc < (D_kcols >> 5); cc++) {
      D1tmp0 = D1_dnVal[offset1 + cid];
      D2tmp0 = D2_dnVal[offset2 + cid];
      multi += D1tmp0 * D2tmp0;
      cid += 32;
    }
    int res = D_kcols & 31;
    D1tmp0 = D2tmp0 = 0;
    if (res) {
      if (off1 < res) {
        D1tmp0 = D1_dnVal[offset1 + cid];
        D2tmp0 = D2_dnVal[offset2 + cid];
      }
      multi += D1tmp0 * D2tmp0;
    }
    for (int stride = 16; stride > 0; stride >>= 1) {
      multi += __shfl_xor_sync(0xffffffff, multi, stride, 32);
    }
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      O_cooVal[eid] = multi;
    }
  }
}

template <typename T>
__global__ void sddmmCOO2Scale(
    int S_mrows,
    int D_kcols,
    int S_nrows,
    const unsigned long Size,
    int* S_cooRowInd,
    int* S_cooColInd,
    T* D1_dnVal_,
    T* D2_dnVal_,
    T* O_cooVal_) {
  int eid = (blockIdx.x << 4) + (threadIdx.y << 2);
  int cid = threadIdx.x << 1;

  int batch_id = blockIdx.y;
  int batch_offset_1 = batch_id * S_mrows * D_kcols;
  int batch_offset_2 = batch_id * S_nrows * D_kcols;
  int batch_offset_out = batch_id * Size;

  T* D1_dnVal = D1_dnVal_ + batch_offset_1;
  T* D2_dnVal = D2_dnVal_ + batch_offset_2;
  T* O_cooVal = O_cooVal_ + batch_offset_out;

  if (blockIdx.x < Size / 16) {
    T multi[4] = {0, 0, 0, 0};
    int offset1[4], offset2[4];
    float2 D1tmp[4], D2tmp[4];
    Load<int4, int>(offset1, S_cooRowInd, eid);
    Load<int4, int>(offset2, S_cooColInd, eid);
    selfMulConst4<int>(offset1, D_kcols);
    selfMulConst4<int>(offset2, D_kcols);

    for (int i = 0; i < (D_kcols >> 5); i++) {
      Load4<float2, float>(D1tmp, D1_dnVal, offset1, cid);
      Load4<float2, float>(D2tmp, D2_dnVal, offset2, cid);
      vec2Dot4<float2>(multi, D1tmp, D2tmp);
      cid += 32;
    }
    int res = D_kcols & 31;
    if (res) {
      int cid2 = threadIdx.x + D_kcols - res;
      float D1[4] = {0, 0, 0, 0}, D2[4] = {0, 0, 0, 0};
      for (int i = 0; i < (res >> 4) + 1; i++) {
        if ((i << 4) + threadIdx.x < res) {
          Load4<float, float>(D1, D1_dnVal, offset1, cid2);
          Load4<float, float>(D2, D2_dnVal, offset2, cid2);
          Dot4<float>(multi, D1, D2);
          cid2 += 16;
        }
      }
    }
    AllReduce4<float>(multi, 8, 32);
    if (threadIdx.x == 0) {
      Store<float4, float>(O_cooVal, multi, eid);
    }
  } else // Dynamic parrallel?
  {
    eid = Size - (Size & 15) + (blockIdx.x - (Size / 16));
    int offset1 = S_cooRowInd[eid] * D_kcols;
    int offset2 = S_cooColInd[eid] * D_kcols;
    T multi = 0;
    int off1 = cid = threadIdx.x << 1;
    float2 D1tmp0, D2tmp0;
    for (int cc = 0; cc < (D_kcols >> 5); cc++) {
      Load<float2, float>(D1tmp0, D1_dnVal, offset1 + cid);
      Load<float2, float>(D2tmp0, D2_dnVal, offset2 + cid);
      multi += vecDot2<float2, float>(D1tmp0, D2tmp0);
      cid += 32;
    }
    int res = D_kcols & 31;
    D1tmp0.x = D1tmp0.y = D2tmp0.x = D2tmp0.y = 0;
    if (res) {
      if (off1 < res) {
        Load<float2, float>(D1tmp0, D1_dnVal, offset1 + cid);
        Load<float2, float>(D2tmp0, D2_dnVal, offset2 + cid);
      }
      multi += vecDot2<float2, float>(D1tmp0, D2tmp0);
    }
    for (int stride = 8; stride > 0; stride >>= 1) {
      multi += __shfl_xor_sync(0xffffffff, multi, stride, 32);
    }
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      O_cooVal[eid] = multi;
    }
  }
}

template <typename T>
__global__ void sddmmCOO1Scale(
    int S_mrows,
    int D_kcols,
    int S_nrows,
    const unsigned long Size,
    int* S_cooRowInd,
    int* S_cooColInd,
    T* D1_dnVal_,
    T* D2_dnVal_,
    T* O_cooVal_) {
  int eid = (blockIdx.x << 4) + (threadIdx.y << 2);
  int cid = threadIdx.x;

  int batch_id = blockIdx.y;
  int batch_offset_1 = batch_id * S_mrows * D_kcols;
  int batch_offset_2 = batch_id * S_nrows * D_kcols;
  int batch_offset_out = batch_id * Size;

  T* D1_dnVal = D1_dnVal_ + batch_offset_1;
  T* D2_dnVal = D2_dnVal_ + batch_offset_2;
  T* O_cooVal = O_cooVal_ + batch_offset_out;

  if (blockIdx.x < Size / 16) {
    T multi[4] = {0, 0, 0, 0};
    int offset1[4], offset2[4];
    float D1tmp[4], D2tmp[4];
    Load<int4, int>(offset1, S_cooRowInd, eid);
    Load<int4, int>(offset2, S_cooColInd, eid);
    selfMulConst4<int>(offset1, D_kcols);
    selfMulConst4<int>(offset2, D_kcols);

    for (int i = 0; i < (D_kcols >> 5); i++) {
      Load4<float, float>(D1tmp, D1_dnVal, offset1, cid);
      Load4<float, float>(D2tmp, D2_dnVal, offset2, cid);
      Dot4<float>(multi, D1tmp, D2tmp);
      cid += 32;
    }
    int res = D_kcols & 31;
    if (res) {
      float D1[4] = {0, 0, 0, 0}, D2[4] = {0, 0, 0, 0};
      if (threadIdx.x < res) {
        Load4<float, float>(D1, D1_dnVal, offset1, cid);
        Load4<float, float>(D2, D2_dnVal, offset2, cid);
        Dot4<float>(multi, D1, D2);
      }
    }
    AllReduce4<float>(multi, 16, 32);
    if (threadIdx.x == 0) {
      Store<float4, float>(O_cooVal, multi, eid);
    }
  } else // Dynamic parrallel?
  {
    eid = Size - (Size & 15) + (blockIdx.x - (Size / 16));
    int offset1 = S_cooRowInd[eid] * D_kcols;
    int offset2 = S_cooColInd[eid] * D_kcols;
    T multi = 0;
    int off1 = cid = threadIdx.x;
    float D1tmp0, D2tmp0;
    for (int cc = 0; cc < (D_kcols >> 5); cc++) {
      D1tmp0 = D1_dnVal[offset1 + cid];
      D2tmp0 = D2_dnVal[offset2 + cid];
      multi += D1tmp0 * D2tmp0;
      cid += 32;
    }
    int res = D_kcols & 31;
    D1tmp0 = D2tmp0 = 0;
    if (res) {
      if (off1 < res) {
        D1tmp0 = D1_dnVal[offset1 + cid];
        D2tmp0 = D2_dnVal[offset2 + cid];
      }
      multi += D1tmp0 * D2tmp0;
    }
    for (int stride = 16; stride > 0; stride >>= 1) {
      multi += __shfl_xor_sync(0xffffffff, multi, stride, 32);
    }
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      O_cooVal[eid] = multi;
    }
  }
}

template <typename T>
__global__ void sddmmCSR2Scale(
    const int S_mrows,
    int D_kcols,
    int S_nrows,
    const unsigned long Size,
    int* S_csrRowPtr,
    int* S_csrColInd,
    T* D1_dnVal_,
    T* D2_dnVal_,
    T* O_csrVal_) {
  int eid = (blockIdx.x << 4) + (threadIdx.y << 2);
  int cid = threadIdx.x << 1;

  int batch_id = blockIdx.y;
  int batch_offset_1 = batch_id * S_mrows * D_kcols;
  int batch_offset_2 = batch_id * S_nrows * D_kcols;
  int batch_offset_out = batch_id * Size;

  T* D1_dnVal = D1_dnVal_ + batch_offset_1;
  T* D2_dnVal = D2_dnVal_ + batch_offset_2;
  T* O_csrVal = O_csrVal_ + batch_offset_out;

  if (blockIdx.x < Size / 16) {
    T multi[4] = {0, 0, 0, 0};
    int offset1[4], offset2[4];
    float2 D1tmp[4], D2tmp[4];
    Load<int4, int>(offset2, S_csrColInd, eid);
    offset1[0] = findRow(S_csrRowPtr, eid, 0, S_mrows);
    offset1[3] = findRow(S_csrRowPtr, eid + 3, offset1[0], S_mrows);
    offset1[1] = findRow(S_csrRowPtr, eid + 1, offset1[0], offset1[3]);
    offset1[2] = findRow(S_csrRowPtr, eid + 2, offset1[1], offset1[3]);
    selfMulConst4<int>(offset1, D_kcols);
    selfMulConst4<int>(offset2, D_kcols);

    for (int i = 0; i < (D_kcols >> 5); i++) {
      Load4<float2, float>(D1tmp, D1_dnVal, offset1, cid);
      Load4<float2, float>(D2tmp, D2_dnVal, offset2, cid);
      vec2Dot4<float2>(multi, D1tmp, D2tmp);
      cid += 32;
    }
    int res = D_kcols & 31;
    if (res) {
      int cid2 = threadIdx.x + D_kcols - res;
      float D1[4] = {0, 0, 0, 0}, D2[4] = {0, 0, 0, 0};
      for (int i = 0; i < (res >> 4) + 1; i++) {
        if ((i << 4) + threadIdx.x < res) {
          Load4<float, float>(D1, D1_dnVal, offset1, cid2);
          Load4<float, float>(D2, D2_dnVal, offset2, cid2);
          Dot4<float>(multi, D1, D2);
          cid2 += 16;
        }
      }
    }
    AllReduce4<float>(multi, 8, 32);
    if (threadIdx.x == 0) {
      Store<float4, float>(O_csrVal, multi, eid);
    }
  } else // Dynamic parrallel?
  {
    eid = Size - (Size & 15) + (blockIdx.x - (Size / 16));
    int offset1 = findRow(S_csrRowPtr, eid, 0, S_mrows) * D_kcols;
    int offset2 = S_csrColInd[eid] * D_kcols;
    T multi = 0;
    int off1 = cid = threadIdx.x << 1;
    float2 D1tmp0, D2tmp0;
    for (int cc = 0; cc < (D_kcols >> 5); cc++) {
      Load<float2, float>(D1tmp0, D1_dnVal, offset1 + cid);
      Load<float2, float>(D2tmp0, D2_dnVal, offset2 + cid);
      multi += vecDot2<float2, float>(D1tmp0, D2tmp0);
      cid += 32;
    }
    int res = D_kcols & 31;
    D1tmp0.x = D1tmp0.y = D2tmp0.x = D2tmp0.y = 0;
    if (res) {
      if (off1 < res) {
        Load<float2, float>(D1tmp0, D1_dnVal, offset1 + cid);
        Load<float2, float>(D2tmp0, D2_dnVal, offset2 + cid);
      }
      multi += vecDot2<float2, float>(D1tmp0, D2tmp0);
    }
    for (int stride = 8; stride > 0; stride >>= 1) {
      multi += __shfl_xor_sync(0xffffffff, multi, stride, 32);
    }
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      O_csrVal[eid] = multi;
    }
  }
}

template <typename T>
__global__ void sddmmCSR1Scale(
    const int S_mrows,
    int D_kcols,
    int S_nrows,
    const unsigned long Size,
    int* S_csrRowPtr,
    int* S_csrColInd,
    T* D1_dnVal_,
    T* D2_dnVal_,
    T* O_csrVal_) {
  int eid = (blockIdx.x << 4) + (threadIdx.y << 2);
  int cid = threadIdx.x;

  int batch_id = blockIdx.y;
  int batch_offset_1 = batch_id * S_mrows * D_kcols;
  int batch_offset_2 = batch_id * S_nrows * D_kcols;
  int batch_offset_out = batch_id * Size;

  T* D1_dnVal = D1_dnVal_ + batch_offset_1;
  T* D2_dnVal = D2_dnVal_ + batch_offset_2;
  T* O_csrVal = O_csrVal_ + batch_offset_out;

  if (blockIdx.x < Size / 16) {
    T multi[4] = {0, 0, 0, 0};
    int offset1[4], offset2[4];
    float D1tmp[4], D2tmp[4];

    Load<int4, int>(offset2, S_csrColInd, eid);

    offset1[0] = findRow(S_csrRowPtr, eid, 0, S_mrows);
    offset1[3] = findRow(S_csrRowPtr, eid + 3, offset1[0], S_mrows);
    offset1[1] = findRow(S_csrRowPtr, eid + 1, offset1[0], offset1[3]);
    offset1[2] = findRow(S_csrRowPtr, eid + 2, offset1[1], offset1[3]);

    selfMulConst4<int>(offset1, D_kcols);
    selfMulConst4<int>(offset2, D_kcols);

    for (int i = 0; i < (D_kcols >> 5); i++) {
      Load4<float, float>(D1tmp, D1_dnVal, offset1, cid);
      Load4<float, float>(D2tmp, D2_dnVal, offset2, cid);
      Dot4<float>(multi, D1tmp, D2tmp);
      cid += 32;
    }
    int res = D_kcols & 31;
    if (res) {
      float D1[4] = {0, 0, 0, 0}, D2[4] = {0, 0, 0, 0};
      if (threadIdx.x < res) {
        Load4<float, float>(D1, D1_dnVal, offset1, cid);
        Load4<float, float>(D2, D2_dnVal, offset2, cid);
        Dot4<float>(multi, D1, D2);
      }
    }
    AllReduce4<float>(multi, 16, 32);
    if (threadIdx.x == 0) {
      Store<float4, float>(O_csrVal, multi, eid);
    }
  } else // Dynamic parrallel?
  {
    eid = Size - (Size & 15) + (blockIdx.x - (Size / 16));
    int offset1 = findRow(S_csrRowPtr, eid, 0, S_mrows) * D_kcols;
    int offset2 = S_csrColInd[eid] * D_kcols;
    T multi = 0;
    int off1 = cid = threadIdx.x;
    float D1tmp0, D2tmp0;
    for (int cc = 0; cc < (D_kcols >> 5); cc++) {
      D1tmp0 = D1_dnVal[offset1 + cid];
      D2tmp0 = D2_dnVal[offset2 + cid];
      multi += D1tmp0 * D2tmp0;
      cid += 32;
    }
    int res = D_kcols & 31;
    D1tmp0 = D2tmp0 = 0;
    if (res) {
      if (off1 < res) {
        D1tmp0 = D1_dnVal[offset1 + cid];
        D2tmp0 = D2_dnVal[offset2 + cid];
      }
      multi += D1tmp0 * D2tmp0;
    }
    for (int stride = 16; stride > 0; stride >>= 1) {
      multi += __shfl_xor_sync(0xffffffff, multi, stride, 32);
    }
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      O_csrVal[eid] = multi;
    }
  }
}

torch::Tensor sddmm_cuda_coo(
    torch::Tensor rowind,
    torch::Tensor colind,
    torch::Tensor D1,
    torch::Tensor D2) {
  const auto batch_size = D1.size(0);
  const auto m = D1.size(1);
  const auto k = D1.size(2);
  const auto n = D2.size(1);
  const auto nnz = rowind.size(0);
  auto out = torch::empty({batch_size, nnz}, D1.options());

  if (out.numel() == 0)
    return out;

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  dim3 grid_dim(nnz / 16 + (nnz & 15), batch_size, 1);
  if ((k % 4) == 0) {
    dim3 block_dim(8, 4, 1);
    sddmmCOO4Scale<<<grid_dim, block_dim, 0, stream>>>(
        m,
        k,
        n,
        nnz,
        rowind.data_ptr<int>(),
        colind.data_ptr<int>(),
        D1.data_ptr<float>(),
        D2.data_ptr<float>(),
        out.data_ptr<float>());
  } else if ((k % 2) == 0) {
    dim3 block_dim(16, 4, 1);
    sddmmCOO2Scale<<<grid_dim, block_dim, 0, stream>>>(
        m,
        k,
        n,
        nnz,
        rowind.data_ptr<int>(),
        colind.data_ptr<int>(),
        D1.data_ptr<float>(),
        D2.data_ptr<float>(),
        out.data_ptr<float>());
  } else {
    dim3 block_dim(32, 4, 1);
    sddmmCOO1Scale<<<grid_dim, block_dim, 0, stream>>>(
        m,
        k,
        n,
        nnz,
        rowind.data_ptr<int>(),
        colind.data_ptr<int>(),
        D1.data_ptr<float>(),
        D2.data_ptr<float>(),
        out.data_ptr<float>());
  }
  AT_CUDA_CHECK(hipGetLastError());
  return out;
}

torch::Tensor sddmm_cuda_csr(
    torch::Tensor rowptr,
    torch::Tensor colind,
    torch::Tensor D1,
    torch::Tensor D2) {
  const auto batch_size = D1.size(0);
  const auto m = D1.size(1);
  const auto k = D1.size(2);
  const auto n = D2.size(1);
  const auto nnz = colind.size(0);
  auto out = torch::empty({batch_size, nnz}, D1.options());

  if (out.numel() == 0)
    return out;

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  dim3 grid_dim(nnz / 16 + (nnz & 15), batch_size, 1);

  if ((k % 2) == 0) {
    dim3 block_dim(16, 4, 1);
    sddmmCSR2Scale<<<grid_dim, block_dim, 0, stream>>>(
        m,
        k,
        n,
        nnz,
        rowptr.data_ptr<int>(),
        colind.data_ptr<int>(),
        D1.data_ptr<float>(),
        D2.data_ptr<float>(),
        out.data_ptr<float>());
  } else {
    dim3 block_dim(32, 4, 1);
    sddmmCSR1Scale<<<grid_dim, block_dim, 0, stream>>>(
        m,
        k,
        n,
        nnz,
        rowptr.data_ptr<int>(),
        colind.data_ptr<int>(),
        D1.data_ptr<float>(),
        D2.data_ptr<float>(),
        out.data_ptr<float>());
  }
  AT_CUDA_CHECK(hipGetLastError());
  return out;
}
} // namespace ge_spmm

torch::Tensor coo_sddmm(
    torch::Tensor D1,
    torch::Tensor D2,
    torch::Tensor unused,
    torch::Tensor rowind,
    torch::Tensor colind) {
  TORCH_CHECK(rowind.device().type() == torch::kCUDA);
  TORCH_CHECK(colind.device().type() == torch::kCUDA);
  TORCH_CHECK(D1.device().type() == torch::kCUDA);
  TORCH_CHECK(D2.device().type() == torch::kCUDA);
  TORCH_CHECK(rowind.is_contiguous());
  TORCH_CHECK(colind.is_contiguous());
  TORCH_CHECK(D1.is_contiguous());
  TORCH_CHECK(D2.is_contiguous());
  TORCH_CHECK(rowind.dtype() == torch::kInt32);
  TORCH_CHECK(colind.dtype() == torch::kInt32);
  TORCH_CHECK(D1.dtype() == torch::kFloat32);
  TORCH_CHECK(D2.dtype() == torch::kFloat32);

  TORCH_CHECK(
      D1.device() == D2.device(), "a should be in the same device as b");
  TORCH_CHECK(
      D1.device() == rowind.device(),
      "a should be in the same device as row_offsets");
  TORCH_CHECK(
      D1.device() == colind.device(),
      "a should be in the same device as column_indices");
  return ge_spmm::sddmm_cuda_coo(rowind, colind, D1, D2);
}

torch::Tensor csr_sddmm(
    torch::Tensor D1,
    torch::Tensor D2,
    torch::Tensor unused,
    torch::Tensor rowptr,
    torch::Tensor colind) {
  TORCH_CHECK(rowptr.device().type() == torch::kCUDA);
  TORCH_CHECK(colind.device().type() == torch::kCUDA);
  TORCH_CHECK(D1.device().type() == torch::kCUDA);
  TORCH_CHECK(D2.device().type() == torch::kCUDA);
  TORCH_CHECK(rowptr.is_contiguous());
  TORCH_CHECK(colind.is_contiguous());
  TORCH_CHECK(D1.is_contiguous());
  TORCH_CHECK(D2.is_contiguous());
  TORCH_CHECK(rowptr.dtype() == torch::kInt32);
  TORCH_CHECK(colind.dtype() == torch::kInt32);
  TORCH_CHECK(D1.dtype() == torch::kFloat32);
  TORCH_CHECK(D2.dtype() == torch::kFloat32);

  TORCH_CHECK(
      D1.device() == D2.device(), "a should be in the same device as b");
  TORCH_CHECK(
      D1.device() == rowptr.device(),
      "a should be in the same device as row_offsets");
  TORCH_CHECK(
      D1.device() == colind.device(),
      "a should be in the same device as column_indices");
  return ge_spmm::sddmm_cuda_csr(rowptr, colind, D1, D2);
}

TORCH_LIBRARY_FRAGMENT(xformers, m) {
  m.def(TORCH_SELECTIVE_SCHEMA(
      "xformers::csr_sddmm(Tensor a, Tensor b, Tensor row_indices, Tensor row_offsets, Tensor column_indices) -> Tensor"));
  m.def(TORCH_SELECTIVE_SCHEMA(
      "xformers::coo_sddmm(Tensor a, Tensor b, Tensor row_indices, Tensor row_offsets, Tensor column_indices) -> Tensor"));
}

TORCH_LIBRARY_IMPL(xformers, CUDA, m) {
  m.impl(TORCH_SELECTIVE_NAME("xformers::csr_sddmm"), TORCH_FN(csr_sddmm));
  m.impl(TORCH_SELECTIVE_NAME("xformers::coo_sddmm"), TORCH_FN(coo_sddmm));
}
