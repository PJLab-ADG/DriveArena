#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
// This file is auto-generated. See "generate_kernels.py"
#ifndef XFORMERS_MEM_EFF_ATTENTION_DISABLE_BACKWARD
#include "../../kernel_backward.h"
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm50, cutlass::half_t, false, false, false, 64, 64, 32>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm50, cutlass::half_t, false, false, false, 64, 64, 32>::kMinBlocksPerSm)
fmha_cutlassB_f16_notaligned_64x64_k32_sm50(typename AttentionBackwardKernel<cutlass::arch::Sm50, cutlass::half_t, false, false, false, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm50, cutlass::half_t, false, false, false, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_f16_notaligned_64x64_k32_sm50` is for sm50-sm70, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm70, cutlass::half_t, false, false, false, 64, 64, 32>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm70, cutlass::half_t, false, false, false, 64, 64, 32>::kMinBlocksPerSm)
fmha_cutlassB_f16_notaligned_64x64_k32_sm70(typename AttentionBackwardKernel<cutlass::arch::Sm70, cutlass::half_t, false, false, false, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 700
#if __CUDA_ARCH__ < 750
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm70, cutlass::half_t, false, false, false, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_f16_notaligned_64x64_k32_sm70` is for sm70-sm75, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm75, cutlass::half_t, false, false, false, 64, 64, 32>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm75, cutlass::half_t, false, false, false, 64, 64, 32>::kMinBlocksPerSm)
fmha_cutlassB_f16_notaligned_64x64_k32_sm75(typename AttentionBackwardKernel<cutlass::arch::Sm75, cutlass::half_t, false, false, false, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750
#if __CUDA_ARCH__ < 800
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm75, cutlass::half_t, false, false, false, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_f16_notaligned_64x64_k32_sm75` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
#endif // XFORMERS_MEM_EFF_ATTENTION_DISABLE_BACKWARD
