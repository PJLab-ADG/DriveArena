#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
// This file is auto-generated. See "generate_kernels.py"
#ifndef XFORMERS_MEM_EFF_ATTENTION_DISABLE_BACKWARD
#include "../../kernel_backward.h"
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm50, float, false, false, false, 64, 64, 65536>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm50, float, false, false, false, 64, 64, 65536>::kMinBlocksPerSm)
fmha_cutlassB_f32_notaligned_64x64_k65536_sm50(typename AttentionBackwardKernel<cutlass::arch::Sm50, float, false, false, false, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm50, float, false, false, false, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_f32_notaligned_64x64_k65536_sm50` is for sm50-sm70, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm70, float, false, false, false, 64, 64, 65536>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm70, float, false, false, false, 64, 64, 65536>::kMinBlocksPerSm)
fmha_cutlassB_f32_notaligned_64x64_k65536_sm70(typename AttentionBackwardKernel<cutlass::arch::Sm70, float, false, false, false, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 700
#if __CUDA_ARCH__ < 750
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm70, float, false, false, false, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_f32_notaligned_64x64_k65536_sm70` is for sm70-sm75, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm75, float, false, false, false, 64, 64, 65536>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm75, float, false, false, false, 64, 64, 65536>::kMinBlocksPerSm)
fmha_cutlassB_f32_notaligned_64x64_k65536_sm75(typename AttentionBackwardKernel<cutlass::arch::Sm75, float, false, false, false, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750
#if __CUDA_ARCH__ < 800
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm75, float, false, false, false, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_f32_notaligned_64x64_k65536_sm75` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
#endif // XFORMERS_MEM_EFF_ATTENTION_DISABLE_BACKWARD
