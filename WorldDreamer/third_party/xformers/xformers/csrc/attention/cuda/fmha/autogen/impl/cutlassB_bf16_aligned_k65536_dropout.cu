#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
// This file is auto-generated. See "generate_kernels.py"
#ifndef XFORMERS_MEM_EFF_ATTENTION_DISABLE_BACKWARD
#include "../../kernel_backward.h"
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, false, 128, 64, 65536>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, false, 128, 64, 65536>::kMinBlocksPerSm)
fmha_cutlassB_bf16_aligned_128x64_k65536_dropout_sm80(typename AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, false, 128, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 900
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, false, 128, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_bf16_aligned_128x64_k65536_dropout_sm80` is for sm80-sm90, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, false, 64, 64, 65536>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, false, 64, 64, 65536>::kMinBlocksPerSm)
fmha_cutlassB_bf16_aligned_64x64_k65536_dropout_sm80(typename AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, false, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 900
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, false, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_bf16_aligned_64x64_k65536_dropout_sm80` is for sm80-sm90, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
#endif // XFORMERS_MEM_EFF_ATTENTION_DISABLE_BACKWARD
